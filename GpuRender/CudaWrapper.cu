#include "hip/hip_runtime.h"
#include ""

#include "CudaWrapper.cuh"

#include <exception>
#include <string>

void* CudaWrapper::CudaMalloc(size_t size)
{
    void* mem;
    auto status = hipMalloc((void**)&mem, size);
    if (status != hipSuccess)
    {
        auto str = hipGetErrorString(status);
        throw std::exception("hipMalloc failed!");
    }
    else
    {
        return mem;
    }
}

void CudaWrapper::CudaMemset(void* memAddress, int value, size_t numBytes)
{
    auto status = hipMemset(memAddress, value, numBytes);
    if (status != hipSuccess)
    {
        throw std::exception("hipMemset failed!");
    }
}

void CudaWrapper::CudaSetDevice(int deviceNum)
{
    auto status = hipSetDevice(deviceNum);
    if (status != hipSuccess)
    {
        throw std::exception("hipSetDevice failed!");
    }
}

void CudaWrapper::CudaMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
    auto status = hipMemcpy(dst, src, count, kind);
    if (status != hipSuccess)
    {
        throw std::exception("hipMemcpy failed!");
    }
}

void CudaWrapper::CudaCheckLaunchErrors()
{
    // Check for any errors launching the kernel
    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        std::string str = "addKernel launch failed: ";
        str += hipGetErrorString(cudaStatus);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}

void CudaWrapper::CudaDeviceSynchronize()
{
    auto cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        throw std::exception("hipDeviceSynchronize returned an error code after launching addKernel!\n");
    }
}