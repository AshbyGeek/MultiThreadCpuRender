#include "hip/hip_runtime.h"
#include ""

#include "CudaWrapper.cuh"

#include <exception>
#include <string>

void* CudaWrapper::CudaMalloc(size_t size)
{
    void* mem;
    auto status = hipMalloc((void**)&mem, size);
    if (status != hipSuccess)
    {
        std::string str = "hipMalloc returned an error: ";
        str += hipGetErrorString(status);
        str += "\n";
        throw new std::exception(str.c_str());
    }
    else
    {
        return mem;
    }
}

void CudaWrapper::CudaMemset(void* memAddress, int value, size_t numBytes)
{
    auto status = hipMemset(memAddress, value, numBytes);
    if (status != hipSuccess)
    {
        std::string str = "hipMemset returned an error: ";
        str += hipGetErrorString(status);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}

void CudaWrapper::CudaSetDevice(int deviceNum)
{
    auto status = hipSetDevice(deviceNum);
    if (status != hipSuccess)
    {
        std::string str = "hipSetDevice returned an error: ";
        str += hipGetErrorString(status);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}

void CudaWrapper::CudaMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
    auto status = hipMemcpy(dst, src, count, kind);
    if (status != hipSuccess)
    {
        std::string str = "hipMemcpy returned an error: ";
        str += hipGetErrorString(status);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}

void CudaWrapper::CudaCheckLaunchErrors()
{
    // Check for any errors launching the kernel
    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        std::string str = "addKernel launch failed: ";
        str += hipGetErrorString(cudaStatus);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}

void CudaWrapper::CudaDeviceSynchronize()
{
    auto cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        std::string str = "hipDeviceSynchronize returned an error: ";
        str += hipGetErrorString(cudaStatus);
        str += "\n";
        throw new std::exception(str.c_str());
    }
}