#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <vector>

#include "Image.cuh"
#include "CudaWrapper.cuh"
#include "CudaRender.cuh"

const int THREADS_PER_BLOCK = 1024;

using namespace CudaWrapper;

struct OpacityResult
{
    __device__
    OpacityResult() {}

    __device__
    OpacityResult(unsigned char value)
    {
        this->value = value;
        isvalid = true;
    }

    bool isvalid = false;
    uint8_t value = 0;
};

__device__
Pixel BlendPixels(Pixel p1, Pixel p2)
{
    Pixel newPixel;
    newPixel.R = (p1.R * (255 - p2.A) + p2.R * p2.A) / 255;
    newPixel.G = (p1.G * (255 - p2.A) + p2.G * p2.A) / 255;
    newPixel.B = (p1.B * (255 - p2.A) + p2.B * p2.A) / 255;
    newPixel.A = p1.A + p2.A*(255 - p1.A);
    return newPixel;
}

__device__
Pixel atomicAlphaMax(Pixel* address, Pixel value)
{
    unsigned int* addrAsUint = (unsigned int*)address;
    unsigned int old = *addrAsUint;
    unsigned int assumed;

    do
    {
        assumed = old;
        Pixel* tmp = (Pixel*)&assumed;
        if (value.A > tmp->A)
        {
            old = atomicCAS(addrAsUint, assumed, *(unsigned int*)&value);
        }
        else
        {
            break;
        }
    } while (assumed != old);
    return *((Pixel*)&old);
}

union IntBytes
{
    int intVal;
    uint8_t bytes[4];
};

__device__
uint8_t atomicMax(uint8_t* address, uint8_t value)
{
    int* fullAddress = (int*)((size_t)address & ~0x3);
    int offset = (size_t)address & 0x3;

    while (true)
    {
        // Get the current value
        IntBytes curVal;
        curVal.intVal = *fullAddress;

        // If the value currently in the memory address is larger than the new value, discard the new value
        if (curVal.bytes[offset] >= value)
        {
            return curVal.bytes[offset];
        }

        IntBytes newVal = curVal;
        newVal.bytes[offset] = value;
        
        // Place the new value into memory if nobody else has changed 
        //   the memory address since we pulled the value
        int oldVal = atomicCAS(fullAddress, curVal.intVal, newVal.intVal);

        // If the swap was successful (nobody else changed the memory location)
        // then we're done
        // otherwise repeat the entire process
        if (curVal.intVal == oldVal)
        {
            return curVal.bytes[offset];
        }
    }
}

__device__
void DrawPixelsAt(uint8_t* opacities, int imgWidth, int imgHeight, Line line, int x, int y, float distFromPixel)
{
    if (x > imgWidth || y > imgHeight)
        return;

    uint8_t val = (1 - abs(distFromPixel)) * 255;
    if (val > 0)
    {
        atomicMax(&opacities[x + y * imgWidth], val);
    }
}

__global__
void DrawLineYCentric(uint8_t* opacities, int imgWidth, int imgHeight, Line line)
{
    Point start = line.start;
    Point end = line.end;
    if (line.start.y > line.end.y)
    {
        start = line.end;
        end = line.start;
    }

    int dx = end.x - start.x;
    int dy = end.y - start.y;


    int y = threadIdx.x + blockIdx.x * blockDim.x;
    if (y > abs(line.Vector().y))
        return;
    y += start.y;

    float x = start.x + dx / (float)dy * (y - start.y);
    if (threadIdx.y == 1)
    {
        DrawPixelsAt(opacities, imgWidth, imgHeight, line, ceil(x), y, x - ceil(x));
    }
    else
    {
        DrawPixelsAt(opacities, imgWidth, imgHeight, line, floor(x), y, x - floor(x));
    }
}

__global__
void DrawLineXCentric(uint8_t* opacities, int imgWidth, int imgHeight, Line line)
{
    Point start = line.start;
    Point end = line.end;
    if (line.start.x > line.end.x)
    {
        start = line.end;
        end = line.start;
    }

    int dx = end.x - start.x;
    int dy = end.y - start.y;

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x > abs(line.Vector().x))
        return;
    x += start.x;

    float y = start.y + dy / (float)dx * (x - start.x);
    if (threadIdx.y == 1)
    {
        DrawPixelsAt(opacities, imgWidth, imgHeight, line, x, ceil(y), y - ceil(y));
    }
    else
    {
        DrawPixelsAt(opacities, imgWidth, imgHeight, line, x, floor(y), y - floor(y));
    }
}

__global__
void FlattenImages(Pixel* base, uint8_t* overlayOpacities, Pixel lineColor, int imgWidth, int imgHeight)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pixelNum = x + y * imgWidth;

    if (x > imgWidth || y > imgHeight)
        return;

    lineColor.A = overlayOpacities[pixelNum];
    base[pixelNum] = BlendPixels(base[pixelNum], lineColor);
}

void CudaRenderImage(Image* image, Pixel color, std::vector<Line>* lines)
{
    Pixel* cudaImg = nullptr;
    uint8_t* cudaOpacities = nullptr;
    try
    {
        CudaSetDevice(0);

        // allocate memory
        int numPixels = image->width * image->height;
                
        hipStream_t memStream;
        hipStreamCreate(&memStream);
        cudaImg = (Pixel*)CudaMalloc(numPixels * sizeof(Pixel));
        hipMemcpyAsync(cudaImg, image->pixels, numPixels * sizeof(Pixel), hipMemcpyHostToDevice, memStream);

        cudaOpacities = (uint8_t*)CudaMalloc(numPixels * sizeof(uint8_t));
        CudaMemset(cudaOpacities, 0, numPixels * sizeof(uint8_t));
        
        for (int i = 0; i < lines->size(); i++)
        {
            auto line = lines->at(i);
            auto lineVect = line.Vector();

            if (abs(lineVect.x) >= abs(lineVect.y))
            {
                dim3 numThreads(THREADS_PER_BLOCK / 2, 2);
                dim3 numBlocks(abs(lineVect.x) / numThreads.x, 1, 1);
                DrawLineXCentric<<<numBlocks,numThreads>>>(cudaOpacities, image->width, image->height, line);
                CudaCheckLaunchErrors();
            }
            else
            {
                dim3 numThreads(THREADS_PER_BLOCK / 2, 2);
                dim3 numBlocks(abs(lineVect.y) / numThreads.x, 1, 1);
                DrawLineYCentric<<<numBlocks,numThreads>>>(cudaOpacities, image->width, image->height, line);
                CudaCheckLaunchErrors();
            }
        }

        CudaDeviceSynchronize();
        
        int sqrtThreadsPerBlock = sqrt(THREADS_PER_BLOCK);
        dim3 numBlocksFlatten(image->width / sqrtThreadsPerBlock, image->height / sqrtThreadsPerBlock);
        dim3 numThreadsFlatten(sqrtThreadsPerBlock, sqrtThreadsPerBlock);
        FlattenImages<<<numBlocksFlatten, numThreadsFlatten>>>(cudaImg, cudaOpacities, color, image->width, image->height);

        // copy the results into the image
        CudaMemcpy(image->pixels, cudaImg, numPixels * sizeof(Pixel), hipMemcpyDeviceToHost);
    }
    catch (const std::exception& ex)
    {
        printf(ex.what());
    }

    // Free allocations
    if (cudaImg != nullptr)
    {
        hipFree(cudaImg);
    }
    if (cudaOpacities != nullptr)
    {
        hipFree(cudaOpacities);
    }
    hipDeviceReset();
}