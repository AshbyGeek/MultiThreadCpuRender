#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <vector>

#include "Image.cuh"
#include "CudaWrapper.cuh"
#include "CudaRender.cuh"

const int THREADS_PER_BLOCK = 1024;

using namespace CudaWrapper;

struct OpacityResult
{
    __device__
    OpacityResult() {}

    __device__
    OpacityResult(unsigned char value)
    {
        this->value = value;
        isvalid = true;
    }

    bool isvalid = false;
    unsigned char value = 0;
};

__device__
Pixel BlendPixels(Pixel p1, Pixel p2)
{
    Pixel newPixel;
    newPixel.R = (p1.R * (255 - p2.A) + p2.R * p2.A) / 255;
    newPixel.G = (p1.G * (255 - p2.A) + p2.G * p2.A) / 255;
    newPixel.B = (p1.B * (255 - p2.A) + p2.B * p2.A) / 255;
    newPixel.A = p1.A + p2.A*(255 - p1.A);
    return newPixel;
}

__device__
OpacityResult PixelOpacity(Point pt, Line line)
{
    float distSquared = line.DistSquaredTo(pt);
    if (abs(distSquared) <= 1)
    {
        auto alpha = round(255 * (1 - abs(distSquared)));
        return OpacityResult(alpha);
    }
    return OpacityResult();
}

__device__
Pixel atomicAlphaMax(Pixel* address, Pixel value)
{
    unsigned int* addrAsUint = (unsigned int*)address;
    unsigned int old = *addrAsUint;
    unsigned int assumed;

    do
    {
        assumed = old;
        Pixel* tmp = (Pixel*)&assumed;
        if (value.A > tmp->A)
        {
            old = atomicCAS(addrAsUint, assumed, *(unsigned int*)&value);
        }
        else
        {
            break;
        }
    } while (assumed != old);
    return *((Pixel*)&old);
}

__device__
Pixel RenderPixel(Point pt, Pixel originalColor, Pixel color, Line* lineArray, int numLines)
{
    OpacityResult maxValue;
    for (int i = 0; i < numLines; i++)
    {
        Line line = lineArray[i];
        auto results = PixelOpacity(pt, line);

        if (results.isvalid && results.value >= maxValue.value)
        {
            maxValue = results;
        }
    }
    
    if (maxValue.isvalid)
    {
        color.A = maxValue.value;
        return BlendPixels(originalColor, color);
    }
    else
    {
        return originalColor;
    }
}

__global__
void RenderKernel(Pixel* pixels, int imgWidth, int imgHeight, Pixel color, Line* lineArray, int numLines)
{
    __shared__ Pixel opacity;
    int pixelNum = blockIdx.x;
    int lineNum = threadIdx.x;
    int x = pixelNum % imgWidth;
    int y = pixelNum / imgWidth;
    
    if (x > imgWidth || y > imgHeight || lineNum > numLines)
        return;

    Point pt;
    pt.x = x;
    pt.y = y;

    Line line = lineArray[lineNum];
    auto results = PixelOpacity(pt, line);

    if (results.isvalid)
    {
        color.A = results.value;
        //auto address = &opacities[pixelNum];
        atomicAlphaMax(&opacity, color);
    }

    __syncthreads();
    if (lineNum == 0)
    {
        pixels[pixelNum] = BlendPixels(pixels[pixelNum], opacity);
    }
}

__global__
void FlattenImages(Pixel* base, Pixel* overlay, int imgWidth, int imgHeight)
{
    int pixelNum = threadIdx.x + blockIdx.x * blockDim.x;
    int x = pixelNum % imgWidth;
    int y = pixelNum / imgWidth;

    if (x > imgWidth || y > imgHeight)
        return;

    base[pixelNum] = BlendPixels(base[pixelNum], overlay[pixelNum]);
}

void CudaRenderImage(Image* image, Pixel color, std::vector<Line>* lines)
{
    Pixel* cudaImg = nullptr;
    Line* cudaLines = nullptr;
    //Pixel* cudaOpacities = nullptr;
    try
    {
        CudaSetDevice(0);

        // allocate memory
        int numPixels = image->width * image->height;
        int numWorkers = numPixels * lines->size();
        
        cudaImg = (Pixel*)CudaMalloc(numPixels * sizeof(Pixel));
        CudaMemcpy(cudaImg, image->pixels, numPixels * sizeof(Pixel), hipMemcpyHostToDevice);

        cudaLines = (Line*)CudaMalloc(lines->size() * sizeof(Line));
        CudaMemcpy(cudaLines, lines->data(), lines->size() * sizeof(Line), hipMemcpyHostToDevice);

        //cudaOpacities = (Pixel*)CudaMalloc(numPixels * sizeof(Pixel));
        //CudaMemset(cudaOpacities, 0, numPixels * sizeof(Pixel));
        
        // Figure out how many threads and blocks
        int numBlocks = numWorkers / THREADS_PER_BLOCK;
        int numThreads = THREADS_PER_BLOCK;

        printf("Number of blocks: %d", numBlocks);

        // launch cuda kernal and wait for it to finish
        RenderKernel<<<numPixels, lines->size()>>>(cudaImg, image->width, image->height, color, cudaLines, lines->size());
        CudaCheckLaunchErrors();
        CudaDeviceSynchronize();

        //numBlocks = numPixels / THREADS_PER_BLOCK;
        //FlattenImages<<<numBlocks, numThreads>>>(cudaImg, cudaOpacities, image->width, image->height);
        //CudaCheckLaunchErrors();
        //CudaDeviceSynchronize();

        // copy the results into the image
        CudaMemcpy(image->pixels, cudaImg, numPixels * sizeof(Pixel), hipMemcpyDeviceToHost);
    }
    catch (const std::exception& ex)
    {
        printf(ex.what());
    }

    // Free allocations
    if (cudaImg != nullptr)
    {
        hipFree(cudaImg);
    }
    if (cudaLines != nullptr)
    {
        hipFree(cudaLines);
    }
    //if (cudaOpacities != nullptr)
    //{
    //    hipFree(cudaOpacities);
    //}
    hipDeviceReset();
}